#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>

#define BLOCK_SIZE 256

// Kernel for parallel reduction using min operation
__global__ void reduceMin(int* input, int* output, int size) {
    __shared__ int sdata[BLOCK_SIZE];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    if (i < size) {
        sdata[tid] = input[i];
    } else {
        sdata[tid] = INT_MAX;
    }

    __syncthreads();

    // Perform reduction within each block
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sdata[tid] = min(sdata[tid], sdata[tid + stride]);
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0) {
        output[blockIdx.x] = sdata[0];
    }
}

// Kernel for parallel reduction using max operation
__global__ void reduceMax(int* input, int* output, int size) {
    __shared__ int sdata[BLOCK_SIZE];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    if (i < size) {
        sdata[tid] = input[i];
    } else {
        sdata[tid] = INT_MIN;
    }

    __syncthreads();

    // Perform reduction within each block
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sdata[tid] = max(sdata[tid], sdata[tid + stride]);
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0) {
        output[blockIdx.x] = sdata[0];
    }
}

// Kernel for parallel reduction using sum operation
__global__ void reduceSum(int* input, int* output, int size) {
    __shared__ int sdata[BLOCK_SIZE];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    if (i < size) {
        sdata[tid] = input[i];
    } else {
        sdata[tid] = 0;
    }

    __syncthreads();

    // Perform reduction within each block
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0) {
        output[blockIdx.x] = sdata[0];
    }
}

// Kernel for parallel reduction using average operation
__global__ void reduceAverage(int* input, float* output, int size) {
    __shared__ float sdata[BLOCK_SIZE];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    if (i < size) {
        sdata[tid] = static_cast<float>(input[i]);
    } else {
        sdata[tid] = 0.0f;
    }

    __syncthreads();

    // Perform reduction within each block
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0) {
        output[blockIdx.x] = sdata[0] / static_cast<float>(size);
    }
}

int main() {
    // Input array
    const int array_size = 256;
    int input[array_size];

    // Initialize input array
    for (int i = 0; i < array_size; ++i) {
        input[i] = i + 1;
    }

    // Allocate device memory
    int* d_input;
    int* d_output_min;
    int* d_output_max;
    int* d_output_sum;
    float* d_output_avg;
    hipMalloc((void**)&d_input, sizeof(int) * array_size);
    hipMalloc((void**)&d_output_min, sizeof(int) * array_size);
    hipMalloc((void**)&d_output_max, sizeof(int) * array_size);
    hipMalloc((void**)&d_output_sum, sizeof(int) * array_size);
    hipMalloc((void**)&d_output_avg, sizeof(float) * array_size);

    // Copy input array to device memory
    hipMemcpy(d_input, input, sizeof(int) * array_size, hipMemcpyHostToDevice);

    // Determine the number of threads and blocks
    int threads_per_block = BLOCK_SIZE;
    int blocks_per_grid = (array_size + threads_per_block - 1) / threads_per_block;

    // Launch the kernels for parallel reduction
    reduceMin<<<blocks_per_grid, threads_per_block>>>(d_input, d_output_min, array_size);
    reduceMax<<<blocks_per_grid, threads_per_block>>>(d_input, d_output_max, array_size);
    reduceSum<<<blocks_per_grid, threads_per_block>>>(d_input, d_output_sum, array_size);
    reduceAverage<<<blocks_per_grid, threads_per_block>>>(d_input, d_output_avg, array_size);

    // Copy the results back to the host
    int min_result, max_result, sum_result;
    float avg_result;
    hipMemcpy(&min_result, d_output_min, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&max_result, d_output_max, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&sum_result, d_output_sum, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&avg_result, d_output_avg, sizeof(float), hipMemcpyDeviceToHost);

    // Print the results
    printf("Minimum value: %d\n", min_result);
    printf("Maximum value: %d\n", max_result);
    printf("Sum: %d\n", sum_result);
    printf("Average: %.2f\n", avg_result);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output_min);
    hipFree(d_output_max);
    hipFree(d_output_sum);
    hipFree(d_output_avg);

    return 0;
}
